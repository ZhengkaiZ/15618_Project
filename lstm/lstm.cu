#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

#include "CycleTimer.h"
#include "lstm.h"

/*************************************************************************
 *
 *  Memory Allocate and Free Functions
 *
 *************************************************************************/

void
allocateModel(Model* model) {
    hipMalloc((void **) &model->W_f, Z * H * sizeof(float));
    hipMalloc((void **) &model->W_i, Z * H * sizeof(float));
    hipMalloc((void **) &model->W_c, Z * H * sizeof(float));
    hipMalloc((void **) &model->W_o, Z * H * sizeof(float));
    hipMalloc((void **) &model->W_y, H * D * sizeof(float));

    hipMalloc((void **) &model->b_f, H * sizeof(float));
    hipMalloc((void **) &model->b_i, H * sizeof(float));
    hipMalloc((void **) &model->b_c, H * sizeof(float));
    hipMalloc((void **) &model->b_o, H * sizeof(float));
    hipMalloc((void **) &model->b_y, D * sizeof(float));
}

void
freeModel(Model *model) {
    hipFree(model->W_f);
    hipFree(model->W_i);
    hipFree(model->W_c);
    hipFree(model->W_o);
    hipFree(model->W_y);

    hipFree(model->b_f);
    hipFree(model->b_i);
    hipFree(model->b_c);
    hipFree(model->b_o);
    hipFree(model->b_y);
}

void
allocateState(State* state) {
    hipMalloc((void **) &state->h, H * sizeof(float));
    hipMalloc((void **) &state->c, H * sizeof(float));
}

void
freeState(State* state) {
    hipFree(state->h);
    hipFree(state->c);
}

void
allocateHiddenState(HiddenState* state) {
    hipMalloc((void **) &state->h_f, H * sizeof(float));
    hipMalloc((void **) &state->h_i, H * sizeof(float));
    hipMalloc((void **) &state->h_c, H * sizeof(float));
    hipMalloc((void **) &state->h_o, H * sizeof(float));

    hipMalloc((void **) &state->X, Z * sizeof(float));
}

void
freeHiddenState(HiddenState* state) {
    hipFree(state->h_f);
    hipFree(state->h_i);
    hipFree(state->h_c);
    hipFree(state->h_o);

    hipFree(state->X);
}

/*************************************************************************
 *
 *  Matrix Functions
 *
 *************************************************************************/

__device__ int
index(int i, int j, int width, int height, bool column_base) {
    if (column_base) {
        return i * width + j;
    } else {
        return j * height + i;
    }
}

__global__ void
matrix_multi(float *x, int x_w, int x_h, bool x_trans, float *y, int y_w, int y_h, bool y_trans, float *result) {
    int index_i = blockIdx.x * blockDim.x + threadIdx.x;
    int index_j = blockIdx.y * blockDim.y + threadIdx.y;

    if (x_trans) {
        int temp = x_h;
        x_h = x_w;
        x_w = temp;
    }

    if (y_trans) {
        int temp = y_h;
        y_h = y_w;
        y_w = temp;
    }

    if (index_i >= x_w || index_j >= y_h)
        return;

    int k;
    int index_z = index_i * x_w + index_j;
    result[index_z] = 0;

    for (k = 0; k < x_h; k++) {
        int index_x = index(index_i, k, x_w, x_h, x_trans);
        int index_y = index(k, index_j, y_w, y_h, y_trans);
        result[index_z] += x[index_x] * y[index_y];
    }
}


// Matrix multi specifically used when 1st matrix x_w is 1.

__global__ void
matrix_multi_single(float *x, float *y, int y_w, int y_h, float *result) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= y_h)
        return;

    int k;
    result[index] = 0;

    for (k = 0; k < y_w; k++) {
        int index_x = k;
        int index_y = k * y_w + index;
        result[index] += x[index_x] * y[index_y];
    }
}

/*************************************************************************
 *
 *  Vector Math Functions
 *
 *************************************************************************/

__global__ void
exp_vector(float *input, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= N)
        return;

    input[index] = exp(input[index]);
}

__global__ void
sigmoid(float *input, float *output, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= N)
        return;

    output[index] = 1 / (1 + exp(-input[index]));
}

__global__ void
dsigmoid(float *input, float *output, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= N)
        return;

    output[index] = input[index] * (1 - input[index]);
}

__global__ void
tanh(float *input, float *output, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= N)
        return;

    output[index] = tanhf(input[index]);
}

__global__ void
dtanh(float *input, float *output, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= N)
        return;

    output[index] = 1 - input[index] * input[index];
}

/*************************************************************************
 *
 *  Point-wise Math Functions
 *
 *************************************************************************/

__global__ void
pointwise_add(float *nums1, float *nums2, float *output, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= N)
        return;

    output[index] = nums1[index] + nums2[index];
}

__global__ void
pointwise_multi(float *nums1, float *nums2, float *output, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= N)
        return;

    output[index] = nums1[index] * nums2[index];
}

/*************************************************************************
 *
 *  Other Math Functions
 *
 *************************************************************************/

__global__ void
devide(float *nums1, float nums2, float *output, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= N)
        return;

    output[index] = nums1[index] / nums2;
}

__global__ void
sum(float* num, float* sum, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= 1)
        return;

    *sum = thrust::reduce(thrust::device, num, num + N);
}

/*************************************************************************
 *
 *  Main Processes
 *
 *************************************************************************/

void
cell_forward(State *old_state, State *state, HiddenState *h, float *prob) {
    dim3 lineDim(256, 1);

    dim3 linesH((H - 1) / lineDim.x + 1);
    dim3 linesD((D - 1) / lineDim.x + 1);

    // Combine input
    hipMemcpy(h->X, old_state->h, H * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(h->X + H, prob, D * sizeof(float), hipMemcpyDeviceToDevice);

    float *temp, *temp2, *temp3;
    hipMalloc((void **) &temp, H * sizeof(float));
    hipMalloc((void **) &temp2, H * sizeof(float));
    hipMalloc((void **) &temp3, D * sizeof(float));

    // Forget Gate
    // hf = sigmoid(X @ Wf + bf)
    matrix_multi_single <<< linesH, lineDim >>> (h->X, model.W_f, Z, H, temp);
    hipDeviceSynchronize();
    pointwise_add <<< linesH, lineDim >>> (temp, model.b_f, temp, H);
    hipDeviceSynchronize();
    sigmoid <<< linesH, lineDim >>> (temp, h->h_f, H);
    hipDeviceSynchronize();

    // Input Gate
    // hi = sigmoid(X @ Wi + bi)
    matrix_multi_single <<< linesH, lineDim >>> (h->X, model.W_i, Z, H, temp);
    hipDeviceSynchronize();
    pointwise_add <<< linesH, lineDim >>> (temp, model.b_i, temp, H);
    hipDeviceSynchronize();
    sigmoid <<< linesH, lineDim >>> (temp, h->h_i, H);
    hipDeviceSynchronize();

    // Detecting input pattern
    // hc = tanh(X @ Wc + bc)
    matrix_multi_single <<< linesH, lineDim >>> (h->X, model.W_c, Z, H, temp);
    hipDeviceSynchronize();
    pointwise_add <<< linesH, lineDim >>> (temp, model.b_c, temp, H);
    hipDeviceSynchronize();
    tanh <<< linesH, lineDim >>> (temp, h->h_c, H);
    hipDeviceSynchronize();

    // Output Gate
    // ho = sigmoid(X @ Wo + bo)
    matrix_multi_single <<< linesH, lineDim >>> (h->X, model.W_o, Z, H, temp);
    hipDeviceSynchronize();
    pointwise_add <<< linesH, lineDim >>> (temp, model.b_o, temp, H);
    hipDeviceSynchronize();
    tanh <<< linesH, lineDim >>> (temp, h->h_o, H);
    hipDeviceSynchronize();

    // c = hf * c_old + hi * hc
    // h = ho * tanh(c)
    pointwise_multi <<< linesH, lineDim >>> (h->h_f, old_state->c, temp, H);
    pointwise_multi <<< linesH, lineDim >>> (h->h_i, h->h_c, temp2, H);
    hipDeviceSynchronize();
    pointwise_add <<< linesH, lineDim >>> (temp, temp2, state->c, H);
    hipDeviceSynchronize();
    tanh <<< linesH, lineDim >>> (state->c, temp, H);
    hipDeviceSynchronize();
    pointwise_multi <<< linesH, lineDim >>> (h->h_o, temp, state->h, H);
    hipDeviceSynchronize();

    // y = h @ Wy + by
    matrix_multi_single <<< linesD, lineDim >>> (state->h, model.W_y, H, D, temp3);
    hipDeviceSynchronize();
    pointwise_add <<< linesD, lineDim >>> (temp3, model.b_y, prob, D);
    hipDeviceSynchronize();

    float sum_exp;
    dim3 singleDim(1, 1);
    dim3 single(1);
    // prob = softmax(y)
    exp_vector <<< linesD, lineDim >>> (prob, D);
    hipDeviceSynchronize();
    sum <<< single, singleDim >>> (prob, &sum_exp, D);
    hipDeviceSynchronize();
    devide <<< linesD, lineDim >>> (prob, sum_exp, prob, D);
    hipDeviceSynchronize();

    hipFree(temp);
    hipFree(temp2);
    hipFree(temp3);
}

void
cell_backward(Model *grad, float *dy, State *old_state, State *state, State *new_state,
                          HiddenState *hiddenState) {
    dim3 lineDim(256, 1);
    dim3 rowDim(1, 256);
    dim3 blockDim(16, 16);

    dim3 lineH((H - 1) / lineDim.x + 1);
    dim3 lineZ((Z - 1) / lineDim.x + 1);
    dim3 rowD(1, (D - 1) / lineDim.x + 1);
    dim3 rowZ(1, (Z - 1) / lineDim.x + 1);
    dim3 blockHD((H-1)/blockDim.x+1, (D-1)/blockDim.y+1);
    dim3 blockZH((Z-1)/blockDim.x+1, (H-1)/blockDim.y+1);

    float *dh_next = new_state->h;
    float *dc_next = new_state->c;

    float *dh, *dc, *temp;
    hipMalloc((void **) &dh, H * sizeof(float));
    hipMalloc((void **) &dc, H * sizeof(float));
    hipMalloc((void **) &temp, H * sizeof(float));

    // Hidden to output gradient
    matrix_multi <<< blockHD, blockDim >>> (state->h, H, 1, false, dy, 1, D, false, grad->W_y);
    hipMemcpy(grad->b_y, dy, D * sizeof(float), hipMemcpyDeviceToDevice);
    hipDeviceSynchronize();
    matrix_multi <<< rowD, rowDim >>> (dy, 1, D, false, model.W_y, H, D, true, dh);
    hipDeviceSynchronize();
    pointwise_add <<< lineH, lineDim >>> (dh, dh_next, dh, H);
    hipDeviceSynchronize();

    // Gradient for h_o in
    // h = h_o * tanh(c)
    float *dho = grad->b_o;
    tanh <<< lineH, lineDim >>> (state->c, dho, H);
    dsigmoid <<< lineH, lineDim >>> (hiddenState->h_o, temp, H);
    hipDeviceSynchronize();
    pointwise_multi <<< lineH, lineDim >>> (dho, dh, dho, H);
    hipDeviceSynchronize();
    pointwise_multi <<< lineH, lineDim >>> (dho, temp, dho, H);
    hipDeviceSynchronize();

    // Gradient for c in
    // h = h_o * tanh(c)
    dtanh <<< lineH, lineDim >>> (state->c, dc, H);
    hipDeviceSynchronize();
    pointwise_multi <<< lineH, lineDim >>> (hiddenState->h_o, dc, dc, H);
    hipDeviceSynchronize();
    pointwise_multi <<< lineH, lineDim >>> (dh, dc, dc, H);
    hipDeviceSynchronize();
    pointwise_add <<< lineH, lineDim >>> (dc_next, dc, dc, H);
    hipDeviceSynchronize();

    // Gradient for h_f in
    // c = h_f * c_old + h_i * h_c
    float *dhf = grad->b_f;
    dsigmoid <<< lineH, lineDim >>> (hiddenState->h_f, dhf, H);
    hipDeviceSynchronize();
    pointwise_multi <<< lineH, lineDim >>> (old_state->c, dhf, dhf, H);
    hipDeviceSynchronize();
    pointwise_multi <<< lineH, lineDim >>> (dc, dhf, dhf, H);
    hipDeviceSynchronize();

    // Gradient for h_i in
    // c = h_f * c_old + h_i * h_c
    float *dhi = grad->b_i;
    dsigmoid <<< lineH, lineDim >>> (hiddenState->h_i, dhi, H);
    hipDeviceSynchronize();
    pointwise_multi <<< lineH, lineDim >>> (hiddenState->h_c, dhi, dhi, H);
    hipDeviceSynchronize();
    pointwise_multi <<< lineH, lineDim >>> (dc, dhi, dhi, H);
    hipDeviceSynchronize();

    // Gradient for h_c in
    // c = h_f * c_old + h_i * h_c
    float *dhc = grad->b_c;
    dtanh <<< lineH, lineDim >>> (hiddenState->h_c, dhc, H);
    hipDeviceSynchronize();
    pointwise_multi <<< lineH, lineDim >>> (hiddenState->h_i, dhc, dhc, H);
    hipDeviceSynchronize();
    pointwise_multi <<< lineH, lineDim >>> (dhc, dc, dhc, H);
    hipDeviceSynchronize();

    // Gate gradients
    matrix_multi <<< blockZH, blockDim >>> (hiddenState->X, Z, 1, false, dhf, 1, H, false, grad->W_f);
    matrix_multi <<< blockZH, blockDim >>> (hiddenState->X, Z, 1, false, dhi, 1, H, false, grad->W_i);
    matrix_multi <<< blockZH, blockDim >>> (hiddenState->X, Z, 1, false, dho, 1, H, false, grad->W_o);
    matrix_multi <<< blockZH, blockDim >>> (hiddenState->X, Z, 1, false, dhc, 1, H, false, grad->W_c);
    hipDeviceSynchronize();

    float *dXf, *dXi, *dXo, *dXc;
    hipMalloc((void **) &dXf, Z * sizeof(float));
    hipMalloc((void **) &dXi, Z * sizeof(float));
    hipMalloc((void **) &dXo, Z * sizeof(float));
    hipMalloc((void **) &dXc, Z * sizeof(float));
    matrix_multi <<< rowZ, rowDim >>> (dhf, 1, H, false, model.W_f, Z, H, true, dXf);
    matrix_multi <<< rowZ, rowDim >>> (dhi, 1, H, false, model.W_i, Z, H, true, dXi);
    matrix_multi <<< rowZ, rowDim >>> (dho, 1, H, false, model.W_o, Z, H, true, dXo);
    matrix_multi <<< rowZ, rowDim >>> (dhc, 1, H, false, model.W_c, Z, H, true, dXc);
    hipDeviceSynchronize();

    float *dX;
    hipMalloc((void **) &dX, Z * sizeof(float));
    pointwise_add <<< lineZ, lineDim >>> (dXf, dXc, dX, Z);
    hipDeviceSynchronize();
    pointwise_add <<< lineZ, lineDim >>> (dX, dXi, dX, Z);
    hipDeviceSynchronize();
    pointwise_add <<< lineZ, lineDim >>> (dX, dXo, dX, Z);
    hipDeviceSynchronize();

    hipMemcpy(dX, dh_next, H * sizeof(float), hipMemcpyDeviceToDevice);
    pointwise_multi <<< lineH, lineDim >>> (hiddenState->h_f, dc, dc_next, H);
    hipDeviceSynchronize();

    hipFree(dXf);
    hipFree(dXi);
    hipFree(dXo);
    hipFree(dXc);
    hipFree(dX);

    hipFree(dh);
    hipFree(dc);
    hipFree(temp);
}

void
train() {
    int input[D], i;
    for (i = 0; i < D; i++) {
        input[i] = i;
    }
    State old_state, state;
    HiddenState hiddenState;
    float *prob;

    double startTime = CycleTimer::currentSeconds();

    allocateModel(&model);
    allocateState(&old_state);
    allocateState(&state);
    allocateHiddenState(&hiddenState);

    hipMalloc((void **) &prob, D * sizeof(float));
    cell_forward(&old_state, &state, &hiddenState, prob);

    Model grad;
    allocateModel(&grad);
    State new_state;
    allocateState(&new_state);

    cell_backward(&grad, prob, &old_state, &state, &new_state, &hiddenState);

    freeModel(&grad);
    freeModel(&model);
    freeState(&old_state);
    freeState(&state);
    freeState(&new_state);
    freeHiddenState(&hiddenState);
    hipFree(prob);

    double endTime = CycleTimer::currentSeconds();

    double overallDuration = endTime - startTime;
    printf("Overall: %.3f ms\t\n", 1000.f * overallDuration);
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
