#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

__device__ int
index(int i, int j, int width, int height, bool column_base) {
    if (column_base) {
        return i * width + j;
    } else {
        return j * height + i;
    }
}

__global__ void
matrix_multi(float* x, int x_w, int x_h, bool x_trans, float* y, int y_w, int y_h, bool y_trans, float* result) {
    int index_i = blockIdx.x * blockDim.x + threadIdx.x;
    int index_j = blockIdx.y * blockDim.y + threadIdx.y;

    if (x_trans) {
        int temp = x_h;
        x_h = x_w;
        x_w = temp;
    }

    if (y_trans) {
        int temp = y_h;
        y_h = y_w;
        y_w = temp;
    }

    if (index_i >= x_w || index_j >= y_h)
        return;

    int i;
    int index_z = index_i * x_w + index_j;
    result[index_z] = 0;

    for (k = 0; k < x_h; k++) {
        int index_x = index(index_i, k, x_w, x_h, x_trans);
        int index_y = index(k, index_j, y_w, y_h, y_trans);
        result[index_z] += x[index_x] * y[index_y]
    }
}

__global__ void
sigmoid(float* input, int N, float* output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= N)
        return;

    output[index] = 1 / (1 + exp(-input[index]))
}

__global__ void
dsigmoid(float* input, int N, float* output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= N)
        return;

    output[index] = input[index] * (1 - input[index])
}

__global__ void
tanh(float* input, int N, float* output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= N)
        return;

    output[index] = tanhf()
}

__global__ void
dtanh(float* input, int N, float* output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= N)
        return;

    output[index] = 1 - input[index] * input[index];
}

void
saxpyCuda(int N, float alpha, float* xarray, float* yarray, float* resultarray) {

    int totalBytes = sizeof(float) * 3 * N;

    // compute number of blocks and threads per block
    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    float* device_x;
    float* device_y;
    float* device_result;

    //
    // allocate device memory buffers on the GPU using hipMalloc
    //
    hipMalloc((void **) &device_x, N * sizeof(float));
    hipMalloc((void **) &device_y, N * sizeof(float));
    hipMalloc((void **) &device_result, N * sizeof(float));

    // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    //
    // copy input arrays to the GPU using hipMemcpy
    //
    hipMemcpy(device_x, xarray, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_y, yarray, N * sizeof(float), hipMemcpyHostToDevice);

    // run kernel
    double startTimeKernel = CycleTimer::currentSeconds();
    saxpy_kernel<<<blocks, threadsPerBlock>>>(N, alpha, device_x, device_y, device_result);
    hipDeviceSynchronize();
    double endTimeKernel = CycleTimer::currentSeconds();

    //
    // copy result from GPU using hipMemcpy
    //
    resultarray = (float *) calloc(N, sizeof(float));
    hipMemcpy(resultarray, device_result, N * sizeof(float), hipMemcpyDeviceToHost);

    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    double overallDurationKernel = endTimeKernel - startTimeKernel;
    printf("Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));
    printf("Kernel Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDurationKernel, toBW(totalBytes, overallDurationKernel));

    // free memory buffers on the GPU
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_result);

}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
